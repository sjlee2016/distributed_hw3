
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#define SIZE 512

__global__ void Multiply(float *A, float *B , float * C){
    //compute thread's row 
    int row = blockIdx.y * blockDim.y + threadIdx.y; 
    //compute thread's column 
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    float s = 0;
    for(int i = 0; i < SIZE; i++){
        s+= A[row*SIZE + i] * B[i*SIZE+col];
    }
    C[row*SIZE+col] = s; // assign result back to C 
}
int main(){
    float A[SIZE*SIZE];
    float B[SIZE*SIZE];
    float C[SIZE*SIZE]; 
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float * d_A, * d_B, *d_C; 
    int BLOCKSIZE;
    printf("Choose block size : ");
    scanf("%d",&BLOCKSIZE);
    int GRIDSIZE = int(SIZE/BLOCKSIZE);
    dim3 dimBlock(BLOCKSIZE,BLOCKSIZE); // 20
    dim3 dimGrid(GRIDSIZE,GRIDSIZE);

    size_t size = SIZE*SIZE*sizeof(float);
    
    for(int i = 0; i < SIZE; i++){
        for(int j = 0; j < SIZE; j++){
            A[i*SIZE+j] = rand()%10;
            B[i*SIZE+j] = rand()%10;
            C[i*SIZE+j] = 0;
        } // 30
    }  

    hipEventRecord(start);
    hipMalloc((void **)&d_A, size);
    hipMemcpy(d_A,A,size,hipMemcpyHostToDevice); 
    hipMalloc((void **)&d_B, size);
    hipMemcpy(d_B,B,size,hipMemcpyHostToDevice); 
    hipMalloc((void **)&d_C, size);
    hipMemcpy(d_C,C,size,hipMemcpyHostToDevice); 
	Multiply<<<dimGrid,dimBlock>>>(d_A, d_B,d_C);
    hipEventRecord(stop);
    hipMemcpy(C, d_C, size, hipMemcpyDeviceToHost);
    hipEventSynchronize(stop);
    float elapsed = 0;
    hipEventElapsedTime(&elapsed, start, stop);
    printf("Using Grid Size [%d, %d] and Block Size [%d, %d]..\n",  dimGrid.x, dimGrid.y,dimBlock.x, dimBlock.y);
    printf("Execution time : %f miliseconds\n", elapsed);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}

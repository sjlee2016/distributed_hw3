
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#define SIZE 1024
#define TILE_WIDTH 32
__global__ void Multiply(float* A, float* B, float* C, int width)
{
    int row = blockDim.y * TILE_WIDTH + threadIdx.y;
    int col = blockDim.x * TILE_WIDTH + threadIdx.x;
    __shared__ float sA[TILE_WIDTH][TILE_WIDTH];
    __shared__ float sB[TILE_WIDTH][TILE_WIDTH];
    int tx = threadIdx.x,ty = threadIdx.y;
    float s = 0;
    for (int i = 0; i < width/TILE_WIDTH; i++) {
    sA[ty][tx] = A[row*width + (i*TILE_WIDTH + tx)];
    sB[ty][tx] = B[col + (i*TILE_WIDTH + ty)*width];
    __syncthreads();
    for (int j = 0; j < TILE_WIDTH; j++)
    s += sA[ty][j] * sB[j][tx];
    __syncthreads();
    }
    C[row*width+col] = s;
}
int main(){
    float A[SIZE*SIZE];
    float B[SIZE*SIZE];
    float C[SIZE*SIZE]; 
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float * d_A, * d_B, *d_C; 
    int BLOCKSIZE;
    printf("Choose block size : ");
    scanf("%d",&BLOCKSIZE);
    int GRIDSIZE = int(SIZE/BLOCKSIZE);
    dim3 dimBlock(BLOCKSIZE,BLOCKSIZE); // 20
    dim3 dimGrid(GRIDSIZE,GRIDSIZE);

    size_t size = SIZE*SIZE*sizeof(float);
    
    for(int i = 0; i < SIZE; i++){
        for(int j = 0; j < SIZE; j++){
            A[i*SIZE+j] = rand()%10;
            B[i*SIZE+j] = rand()%10;
            C[i*SIZE+j] = 0;
        } // 30
    }  

    hipEventRecord(start);
    hipMalloc((void **)&d_A, size);
    hipMemcpy(d_A,A,size,hipMemcpyHostToDevice); 
    hipMalloc((void **)&d_B, size);
    hipMemcpy(d_B,B,size,hipMemcpyHostToDevice); 
    hipMalloc((void **)&d_C, size);
    hipMemcpy(d_C,C,size,hipMemcpyHostToDevice); 
	Multiply<<<dimGrid,dimBlock>>>(d_A, d_B,d_C,64);
    hipEventRecord(stop);
    hipMemcpy(C, d_C, size, hipMemcpyDeviceToHost);
    hipEventSynchronize(stop);
    float elapsed = 0;
    hipEventElapsedTime(&elapsed, start, stop);
    printf("Using Grid Size [%d, %d] and Block Size [%d, %d]..\n",  dimGrid.x, dimGrid.y,dimBlock.x, dimBlock.y);
    printf("Execution time : %f miliseconds\n", elapsed);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}
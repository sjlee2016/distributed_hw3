#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#define SIZE 300000
#define BLOCK_SIZE 512
__global__ void reduction(int *A, int *B){
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    __shared__ int data[BLOCK_SIZE];
    data[threadIdx.x] = A[tid]; // load data to shared memory 
    __syncthreads();
    // iterate of log base 2 block dimension. using stride of 2 
    for(int i = blockDim.x/2; i>0; i>>=1){
        if(threadIdx.x < i){
            if(data[threadIdx.x] < data[threadIdx.x + i]){
                data[threadIdx.x] = data[threadIdx.x + i];
            }
        }
        __syncthreads();
    }
    __syncthreads();
    // thread 0 should write the maximum value to main memory 
    if(threadIdx.x == 0 )
        B[blockIdx.x] = data[0]; 
}
int main(){
    int A[SIZE];
    int * B;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    int * d_A, * d_B; 
	srand(time(NULL));
    size_t size = SIZE*sizeof(int);
    int GRIDSIZE = SIZE / (BLOCK_SIZE<<1);
	if (GRIDSIZE % (BLOCK_SIZE<<1)) 
		GRIDSIZE++;
	B = (int *) malloc(sizeof(int)*GRIDSIZE); 
    dim3 dimBlock(BLOCK_SIZE,1,1); 
    dim3 dimGrid(GRIDSIZE,1,1);

    for(int i = 0; i < SIZE; i++){
            A[i] = rand()%10000;
            if(i<GRIDSIZE)
                B[i] = 0; 
    }  
    hipEventRecord(start);
    hipMalloc((void **)&d_A, size);
    hipMemcpy(d_A,A,size,hipMemcpyHostToDevice); 
    hipMalloc((void **)&d_B, GRIDSIZE*sizeof(int));
	reduction<<<dimGrid,dimBlock>>>(d_A, d_B);
    hipEventRecord(stop);
    hipMemcpy(B, d_B, GRIDSIZE*sizeof(int), hipMemcpyDeviceToHost);
    for(int i = 1; i < GRIDSIZE; i++){
        if(B[0] < B[i])
            B[0] = B[i]; 
    }
    hipEventSynchronize(stop);
    float elapsed = 0;
    hipEventElapsedTime(&elapsed, start, stop);
    printf("Using Grid Size [%d, %d] and Block Size [%d, %d]..\n",  dimGrid.x, dimGrid.y,dimBlock.x, dimBlock.y);
    printf("maximum : %d\n", B[0]); 
    printf("Execution time : %f ms\n", elapsed);
    hipFree(d_A);
    hipFree(d_B);
}

#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#define SIZE 300000
#define BLOCK_SIZE 128
__global__ void reduction(int *A, int *B){
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    B[threadIdx.x] = A[tid]; // init max value 
    __syncthreads();
    // iterate of log base 2 block dimension. using stride of 2 
    for(int i = 1; i< blockDim.x; i *= 2){
        if(threadIdx.x%(2*i) == 0){
            if(B[threadIdx.x] < B[threadIdx.x + i]){
                B[threadIdx.x] = B[threadIdx.x + i];
            }
        }
        __syncthreads();
    }
}
int main(){
    int A[SIZE];
    int * B;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
	srand(time(NULL));
    int * d_A, * d_B; 
    size_t size = SIZE*sizeof(int);
    int GRIDSIZE = SIZE / (BLOCK_SIZE<<1);
	if (GRIDSIZE % (BLOCK_SIZE<<1)) 
		GRIDSIZE++;
	B = (int *) malloc(sizeof(int)*GRIDSIZE); 
    dim3 dimBlock(BLOCK_SIZE,1,1); 
    dim3 dimGrid(GRIDSIZE,1,1);

    for(int i = 0; i < SIZE; i++){
            A[i] = rand()%10000;
            if(i<GRIDSIZE)
                B[i] = 0; 
    }  
    hipEventRecord(start);
    
    hipMalloc((void **)&d_A, size);
    hipMemcpy(d_A,A,size,hipMemcpyHostToDevice); 
    hipMalloc((void **)&d_B, GRIDSIZE*sizeof(int));
	reduction<<<dimGrid,dimBlock>>>(d_A, d_B);
    hipEventRecord(stop);
    hipMemcpy(B, d_B, GRIDSIZE*sizeof(int), hipMemcpyDeviceToHost);
    for(int i = 1; i < GRIDSIZE; i++){
        if(B[0] < B[i])
            B[0] = B[i]; 
    }
    hipEventSynchronize(stop);
    float elapsed = 0;
    hipEventElapsedTime(&elapsed, start, stop);
    printf("Using Grid Size [%d, %d] and Block Size [%d, %d]..\n",  dimGrid.x, dimGrid.y,dimBlock.x, dimBlock.y);
    printf("maximum : %d\n", B[0]); 
    printf("Execution time : %f ms\n", elapsed);
    hipFree(d_A);
    hipFree(d_B);
}
